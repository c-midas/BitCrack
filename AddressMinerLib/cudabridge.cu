#include "hip/hip_runtime.h"
#include "cudabridge.h"


__global__ void addressMinerKernel(int points, unsigned int flags, unsigned int *x, unsigned int *y, unsigned int *chain, unsigned int *numResults, void *results);


void callAddressMinerKernel(KernelParams &params)
{
	addressMinerKernel<<<params.blocks, params.threads>>>(params.points, params.flags, params.x, params.y, params.chain, params.numResults, params.results);
	waitForKernel();
}


void waitForKernel()
{
	// Wait for kernel to complete
	hipError_t err = hipDeviceSynchronize();
	fflush(stdout);
	if(err != hipSuccess) {
		throw CudaException(err);
	}
}