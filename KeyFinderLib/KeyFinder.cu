#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>

#include "ptx.cuh"
#include "secp256k1.cuh"

#include "sha256.cuh"
#include "ripemd160.cuh"

#include "secp256k1.h"
#include "DeviceContextShared.h"

__constant__ unsigned int _TARGET_HASH[5];
__constant__ unsigned int _INC_X[8];
__constant__ unsigned int _INC_Y[8];

static const unsigned int _RIPEMD160_IV_HOST[5] = {
	0x67452301,
	0xefcdab89,
	0x98badcfe,
	0x10325476,
	0xc3d2e1f0
};

static unsigned int swp(unsigned int x)
{
	return (x << 24) | ((x << 8) & 0x00ff0000) | ((x >> 8) & 0x0000ff00) | (x >> 24);
}

hipError_t setTargetHash(const unsigned int hash[5])
{
	unsigned int h[5];


	// Undo the final round of RIPEMD160 and endian swap to save some computation
	for(int i = 0; i < 5; i++) {
		h[i] = swp(hash[i]) - _RIPEMD160_IV_HOST[(i + 1) % 5];
	}

	return hipMemcpyToSymbol(HIP_SYMBOL(_TARGET_HASH), h, sizeof(unsigned int) * 5);
}

hipError_t setIncrementorPoint(const secp256k1::uint256 &x, const secp256k1::uint256 &y)
{
	unsigned int xWords[8];
	unsigned int yWords[8];

	x.exportWords(xWords, 8, secp256k1::uint256::BigEndian);
	y.exportWords(yWords, 8, secp256k1::uint256::BigEndian);

	hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_INC_X), xWords, sizeof(unsigned int) * 8);
	if(err) {
		return err;
	}

	return hipMemcpyToSymbol(HIP_SYMBOL(_INC_Y), yWords, sizeof(unsigned int) * 8);
}

__device__ void hashPublicKey(const unsigned int *x, const unsigned int *y, unsigned int *digestOut)
{
	unsigned int hash[8];

	sha256PublicKey(x, y, hash);

	// Swap to little-endian
	for(int i = 0; i < 8; i++) {
		hash[i] = endian(hash[i]);
	}

	ripemd160sha256NoFinal(hash, digestOut);
}

__device__ void hashPublicKeyCompressed(const unsigned int *x, const unsigned int *y, unsigned int *digestOut)
{
	unsigned int hash[8];

	sha256PublicKeyCompressed(x, y, hash);

	// Swap to little-endian
	for(int i = 0; i < 8; i++) {
		hash[i] = endian(hash[i]);
	}

	ripemd160sha256NoFinal(hash, digestOut);
}

__device__ void addResult(unsigned int *numResultsPtr, void *results, void *info, int size)
{
	grabLock();

	unsigned char *ptr = (unsigned char *)results + (*numResultsPtr);

	memcpy(ptr, info, size);

	(*numResultsPtr)++;
	releaseLock();
}

__device__ void setResultFound(unsigned int *numResultsPtr, void *results, int idx, bool compressed, unsigned int x[8], unsigned int y[8], unsigned int digest[5])
{
	struct KeyFinderDeviceResult r;

	r.block = blockIdx.x;
	r.thread = threadIdx.x;
	r.idx = idx;
	r.compressed = compressed;

	for(int i = 0; i < 8; i++) {
		r.x[i] = x[i];
		r.y[i] = y[i];
	}

	for(int i = 0; i < 5; i++) {
		r.digest[i] = endian(digest[i] + _RIPEMD160_IV[(i + 1) % 5]);
	}
	addResult(numResultsPtr, results, &r, sizeof(r));
}

__device__ bool checkHash(unsigned int hash[5])
{
	bool equal = true;

	for(int i = 0; i < 5; i++) {
		equal &= (hash[i] == _TARGET_HASH[i]);
	}

	return equal;
}

__device__ void doIteration(unsigned int *xPtr, unsigned int *yPtr, unsigned int *chain, int pointsPerThread, unsigned int *numResults, void *results, int compression)
{
	// Multiply together all (_Gx - x) and then invert
	unsigned int inverse[8] = { 0,0,0,0,0,0,0,1 };
	for(int i = 0; i < pointsPerThread; i++) {
		unsigned int x[8];
		unsigned int y[8];
		unsigned int digest[5];

		readInt(xPtr, i, x);
		readInt(yPtr, i, y);

		if(compression == PointCompressionType::UNCOMPRESSED || compression == PointCompressionType::BOTH) {
			hashPublicKey(x, y, digest);

			if(checkHash(digest)) {
				setResultFound(numResults, results, i, false, x, y, digest);
			}
		}

		if(compression == PointCompressionType::COMPRESSED || compression == PointCompressionType::BOTH) {
			hashPublicKeyCompressed(x, y, digest);

			if(checkHash(digest)) {
				setResultFound(numResults, results, i, true, x, y, digest);
			}
		}

		beginBatchAdd(_INC_X, xPtr, chain, i, inverse);
	}

	doBatchInverse(inverse);

	for(int i = pointsPerThread - 1; i >= 0; i--) {

		unsigned int newX[8];
		unsigned int newY[8];

		completeBatchAdd(_INC_X, _INC_Y, xPtr, yPtr, i, chain, inverse, newX, newY);

		writeInt(xPtr, i, newX);
		writeInt(yPtr, i, newY);
	}
}

__device__ void doIterationWithDouble(unsigned int *xPtr, unsigned int *yPtr, unsigned int *chain, int pointsPerThread, unsigned int *numResults, void *results, int compression)
{
	// Multiply together all (_Gx - x) and then invert
	unsigned int inverse[8] = { 0,0,0,0,0,0,0,1 };
	for(int i = 0; i < pointsPerThread; i++) {
		unsigned int x[8];
		unsigned int y[8];
		unsigned int digest[5];

		readInt(xPtr, i, x);
		readInt(yPtr, i, y);

		// uncompressed
		if(compression == 1 || compression == 2) {
			hashPublicKey(x, y, digest);

			if(checkHash(digest)) {
				setResultFound(numResults, results, i, false, x, y, digest);
			}
		}

		// compressed
		if(compression == 0 || compression == 2) {
			hashPublicKeyCompressed(x, y, digest);

			if(checkHash(digest)) {
				setResultFound(numResults, results, i, true, x, y, digest);
			}
		}

		beginBatchAddWithDouble(_INC_X, _INC_Y, xPtr, chain, i, inverse);
	}

	doBatchInverse(inverse);

	for(int i = pointsPerThread - 1; i >= 0; i--) {

		unsigned int newX[8];
		unsigned int newY[8];

		completeBatchAddWithDouble(_INC_X, _INC_Y, xPtr, yPtr, i, chain, inverse, newX, newY);

		writeInt(xPtr, i, newX);
		writeInt(yPtr, i, newY);
	}
}

/**
* Performs a single iteration
*/
__global__ void keyFinderKernel(int points, unsigned int *x, unsigned int *y, unsigned int *chain, unsigned int *numResults, void *results, int compression)
{
	doIteration(x, y, chain, points, numResults, results, compression);
}

__global__ void keyFinderKernelWithDouble(int points, unsigned int *x, unsigned int *y, unsigned int *chain, unsigned int *numResults, void *results, int compression)
{
	doIterationWithDouble(x, y, chain, points, numResults, results, compression);
}