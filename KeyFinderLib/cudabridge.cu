#include "hip/hip_runtime.h"
#include "cudabridge.h"


__global__ void keyFinderKernel(int points, int compression);
__global__ void keyFinderKernelWithDouble(int points, int compression);

void callKeyFinderKernel(KernelParams &params, bool useDouble, int compression)
{
	if(useDouble) {
		keyFinderKernelWithDouble <<<params.blocks, params.threads >> >(params.points, compression);
	} else {
		keyFinderKernel <<<params.blocks, params.threads >> > (params.points, compression);
	}
	waitForKernel();
}


void waitForKernel()
{
    // Check for kernel launch error
    hipError_t err = hipGetLastError();

    if(err != hipSuccess) {
        throw cuda::CudaException(err);
    }
 
    // Wait for kernel to complete
    err = hipDeviceSynchronize();
	fflush(stdout);
	if(err != hipSuccess) {
		throw cuda::CudaException(err);
	}
}